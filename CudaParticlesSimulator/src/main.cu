#include "hip/hip_runtime.h"
#include "memutils.cuh"
#include "cudaParticleSimulator.cuh"

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <iostream>
#include <fstream>
#include <cstdlib>

#ifndef PARTICLE_NUM
    #define PARTICLE_NUM 10000
#endif

#ifndef THREADS_PER_BLOCK
    #define THREADS_PER_BLOCK 128
#endif

#ifndef DURATION
    #define DURATION 10
#endif



int main(int argc, char** argv) {
    // TODO: manage arguments
    if (argc != 3) {
        cerr << "Usage: " << argv[0] << " <version> <time_file>\n";
        return 1;
    }

    // Read the version and time file
    int version = atoi(argv[1]);
    char *time_file = argv[2];

    ofstream time_stream(time_file, ios::out | ios::app);
    if (!time_stream.is_open()) {
        cerr << "Could not open time file: " << time_file << endl;
        return 1;
    }

    // host memory
    double *h_masses = (double*) malloc(PARTICLE_NUM * sizeof(double));

    // TODO: use float3
    double *h_x_pos = (double*) malloc(PARTICLE_NUM * sizeof(double));
    double *h_y_pos = (double*) malloc(PARTICLE_NUM * sizeof(double));
    double *h_z_pos = (double*) malloc(PARTICLE_NUM * sizeof(double));

    double *h_x_vel = (double*) malloc(PARTICLE_NUM * sizeof(double));
    double *h_y_vel = (double*) malloc(PARTICLE_NUM * sizeof(double));
    double *h_z_vel = (double*) malloc(PARTICLE_NUM * sizeof(double));


    // initialize host memory
    int unused;
    cin >> unused;
    cin >> unused;

    for (int i = 0; i < PARTICLE_NUM; i++) {
        cin >> h_x_pos[i] >> h_y_pos[i] >> h_z_pos[i]
            >> h_x_vel[i] >> h_y_vel[i] >> h_z_vel[i]
            >> h_masses[i];
    }

    // allocate and initialize device memory
    // TODO: declare __constant__
    double *d_masses = allocateAndCopy(h_masses, PARTICLE_NUM);

    double *d_x_pos_old = allocateAndCopy(h_x_pos, PARTICLE_NUM);
    double *d_y_pos_old = allocateAndCopy(h_y_pos, PARTICLE_NUM);
    double *d_z_pos_old = allocateAndCopy(h_z_pos, PARTICLE_NUM);

    double *d_x_vel_old = allocateAndCopy(h_x_vel, PARTICLE_NUM); 
    double *d_y_vel_old = allocateAndCopy(h_y_vel, PARTICLE_NUM);
    double *d_z_vel_old = allocateAndCopy(h_z_vel, PARTICLE_NUM);

    double *d_x_pos_new = allocateAndNull(PARTICLE_NUM);
    double *d_y_pos_new = allocateAndNull(PARTICLE_NUM);
    double *d_z_pos_new = allocateAndNull(PARTICLE_NUM);

    double *d_x_vel_new = allocateAndNull(PARTICLE_NUM); 
    double *d_y_vel_new = allocateAndNull(PARTICLE_NUM);
    double *d_z_vel_new = allocateAndNull(PARTICLE_NUM);

    double *d_x_acc = allocateAndNull(PARTICLE_NUM);
    double *d_y_acc = allocateAndNull(PARTICLE_NUM);
    double *d_z_acc = allocateAndNull(PARTICLE_NUM);

    // copy masses from host to device
    hipError_t result = hipMemcpy(d_masses, h_masses, sizeof(double) * PARTICLE_NUM,
        hipMemcpyHostToDevice);
    if (result != hipSuccess) {
        cerr << "Could not copy the masses array to the device \n";
        return 0;
    }

    // copy positions and velocities from host to device
    result = hipMemcpy(d_x_pos_old, h_x_pos, sizeof(double) * PARTICLE_NUM,
        hipMemcpyHostToDevice);
    if (result != hipSuccess) {
        cerr << "Could not copy the x_pos array to the device \n";
        return 0;
    }

    result = hipMemcpy(d_y_pos_old, h_y_pos, sizeof(double) * PARTICLE_NUM,
    hipMemcpyHostToDevice);
    if (result != hipSuccess) {
        cerr << "Could not copy the y_pos array to the device \n";
        return 0;
    }

    result = hipMemcpy(d_z_pos_old, h_z_pos, sizeof(double) * PARTICLE_NUM,
    hipMemcpyHostToDevice);
    if (result != hipSuccess) {
        cerr << "Could not copy the z_pos array to the device \n";
        return 0;
    }

    result = hipMemcpy(d_x_vel_old, h_x_vel, sizeof(double) * PARTICLE_NUM,
    hipMemcpyHostToDevice);
    if (result != hipSuccess) {
        cerr << "Could not copy the x_vel array to the device \n";
        return 0;
    }

    result = hipMemcpy(d_y_vel_old, h_y_vel, sizeof(double) * PARTICLE_NUM,
    hipMemcpyHostToDevice);
    if (result != hipSuccess) {
        cerr << "Could not copy the y_vel array to the device \n";
        return 0;
    }

    result = hipMemcpy(d_z_vel_old, h_z_vel, sizeof(double) * PARTICLE_NUM,
    hipMemcpyHostToDevice);
    if (result != hipSuccess) {
        cerr << "Could not copy the z_vel array to the device \n";
        return 0;
    }

    //set up Cuda Event for timing
    float milliseconds;
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    for (int iter = 0; iter < DURATION; iter++) {
        hipEventRecord(start);
        // set up the kernel launch parameters
        newState << <(PARTICLE_NUM + THREADS_PER_BLOCK - 1) / THREADS_PER_BLOCK, THREADS_PER_BLOCK >> >(
            PARTICLE_NUM,
            d_masses,
            d_x_pos_old,
            d_y_pos_old,
            d_z_pos_old,
            d_x_vel_old,
            d_y_vel_old,
            d_z_vel_old,
            d_x_pos_new,
            d_y_pos_new,
            d_z_pos_new,
            d_x_vel_new,
            d_y_vel_new,
            d_z_vel_new,
            d_x_acc,
            d_y_acc,
            d_z_acc
        );
        result = hipDeviceSynchronize();

        hipEventRecord(stop);
        hipEventSynchronize(stop);

        hipEventElapsedTime(&milliseconds, start, stop);
        if (result != hipSuccess) {
            cerr << "Kernel launch failed with error: " << hipGetErrorString(result) << endl;
            return 0;
        }
        // print the time taken for this iteration
        time_stream << version << " " << THREADS_PER_BLOCK << " " << PARTICLE_NUM << " " << iter << ": " << milliseconds << "ms" << endl;

        // log results
        result = hipMemcpy(h_x_pos, d_x_pos_new, sizeof(double) * PARTICLE_NUM,
            hipMemcpyDeviceToHost);
        if (result != hipSuccess) {
            cerr << "Could not copy the x_pos array to the host \n";
            return 0;
        }
        result = hipMemcpy(h_y_pos, d_y_pos_new, sizeof(double) * PARTICLE_NUM,
            hipMemcpyDeviceToHost);
        if (result != hipSuccess) {
            cerr << "Could not copy the y_pos array to the host \n";
            return 0;
        }
        result = hipMemcpy(h_z_pos, d_z_pos_new, sizeof(double) * PARTICLE_NUM,
            hipMemcpyDeviceToHost);
        if (result != hipSuccess) {
            cerr << "Could not copy the z_pos array to the host \n";
            return 0;
        }

        for (int particle = 0; particle < PARTICLE_NUM; particle++) {
            cout << h_x_pos[particle] << " "
                << h_y_pos[particle] << " "
                << h_z_pos[particle] << " "
                << endl;
        }
        cout << endl;
        // also velocities?

        // swap the old and new positions and velocities
        swap(d_x_pos_old, d_x_pos_new);
        swap(d_y_pos_old, d_y_pos_new);
        swap(d_z_pos_old, d_z_pos_new);

        swap(d_x_vel_old, d_x_vel_new);
        swap(d_y_vel_old, d_y_vel_new);
        swap(d_z_vel_old, d_z_vel_new);
    }

    // Free memory
    free(h_masses);

    free(h_x_pos);
    free(h_y_pos);
    free(h_z_pos);

    free(h_x_vel);
    free(h_y_vel);
    free(h_z_vel);

    // free device memory
    hipFree(d_masses);

    hipFree(d_x_pos_old);
    hipFree(d_y_pos_old);
    hipFree(d_z_pos_old);

    hipFree(d_x_vel_old);
    hipFree(d_y_vel_old);
    hipFree(d_z_vel_old);

    hipFree(d_x_pos_new);
    hipFree(d_y_pos_new);
    hipFree(d_z_pos_new);

    hipFree(d_x_vel_new);
    hipFree(d_y_vel_new);
    hipFree(d_z_vel_new);

    hipFree(d_x_acc);
    hipFree(d_y_acc);
    hipFree(d_z_acc);

    // destroy Cuda Event
    hipEventDestroy(start);
    hipEventDestroy(stop);
    time_stream.close();
    return 0;
}