#include "hip/hip_runtime.h"
#include "memutils.cuh"
#include "cudaParticleSimulator.cuh"

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <iostream>
#include <cstdlib>

#ifndef PARTICLE_NUM
    #define PARTICLE_NUM 10000
#endif

#ifndef THREADS_PER_BLOCK
    #define THREADS_PER_BLOCK 128
#endif

#ifndef DURATION
    #define DURATION 10
#endif



int main(int argc, char** argv) {
    // TODO: manage arguments

    // host memory
    double *h_masses = (double*) malloc(PARTICLE_NUM * sizeof(double));

    // TODO: use float3
    double *h_x_pos = (double*) malloc(PARTICLE_NUM * sizeof(double));
    double *h_y_pos = (double*) malloc(PARTICLE_NUM * sizeof(double));
    double *h_z_pos = (double*) malloc(PARTICLE_NUM * sizeof(double));

    double *h_x_vel = (double*) malloc(PARTICLE_NUM * sizeof(double));
    double *h_y_vel = (double*) malloc(PARTICLE_NUM * sizeof(double));
    double *h_z_vel = (double*) malloc(PARTICLE_NUM * sizeof(double));


    // initialize host memory
    int unused;
    cin >> unused;
    cin >> unused;

    for (int i = 0; i < PARTICLE_NUM; i++) {
        cin >> h_x_pos[i] >> h_y_pos[i] >> h_z_pos[i]
            >> h_x_vel[i] >> h_y_vel[i] >> h_z_vel[i]
            >> h_masses[i];
    }

    // allocate and initialize device memory
    // TODO: declare __constant__
    double *d_masses = allocateAndCopy(h_masses, PARTICLE_NUM);

    double *d_x_pos_old = allocateAndCopy(h_x_pos, PARTICLE_NUM);
    double *d_y_pos_old = allocateAndCopy(h_y_pos, PARTICLE_NUM);
    double *d_z_pos_old = allocateAndCopy(h_z_pos, PARTICLE_NUM);

    double *d_x_vel_old = allocateAndCopy(h_x_vel, PARTICLE_NUM); 
    double *d_y_vel_old = allocateAndCopy(h_y_vel, PARTICLE_NUM);
    double *d_z_vel_old = allocateAndCopy(h_z_vel, PARTICLE_NUM);

    double *d_x_pos_new = allocateAndNull(PARTICLE_NUM);
    double *d_y_pos_new = allocateAndNull(PARTICLE_NUM);
    double *d_z_pos_new = allocateAndNull(PARTICLE_NUM);

    double *d_x_vel_new = allocateAndNull(PARTICLE_NUM); 
    double *d_y_vel_new = allocateAndNull(PARTICLE_NUM);
    double *d_z_vel_new = allocateAndNull(PARTICLE_NUM);

    double *d_x_acc = allocateAndNull(PARTICLE_NUM);
    double *d_y_acc = allocateAndNull(PARTICLE_NUM);
    double *d_z_acc = allocateAndNull(PARTICLE_NUM);

    // copy masses from host to device
    hipError_t result = hipMemcpy(d_masses, h_masses, sizeof(double) * PARTICLE_NUM,
        hipMemcpyHostToDevice);
    if (result != hipSuccess) {
        cerr << "Could not copy the masses array to the device \n";
        return 0;
    }

    // copy positions and velocities from host to device
    result = hipMemcpy(d_x_pos_old, h_x_pos, sizeof(double) * PARTICLE_NUM,
        hipMemcpyHostToDevice);
    if (result != hipSuccess) {
        cerr << "Could not copy the x_pos array to the device \n";
        return 0;
    }

    result = hipMemcpy(d_y_pos_old, h_y_pos, sizeof(double) * PARTICLE_NUM,
    hipMemcpyHostToDevice);
    if (result != hipSuccess) {
        cerr << "Could not copy the y_pos array to the device \n";
        return 0;
    }

    result = hipMemcpy(d_z_pos_old, h_z_pos, sizeof(double) * PARTICLE_NUM,
    hipMemcpyHostToDevice);
    if (result != hipSuccess) {
        cerr << "Could not copy the z_pos array to the device \n";
        return 0;
    }

    result = hipMemcpy(d_x_vel_old, h_x_vel, sizeof(double) * PARTICLE_NUM,
    hipMemcpyHostToDevice);
    if (result != hipSuccess) {
        cerr << "Could not copy the x_vel array to the device \n";
        return 0;
    }

    result = hipMemcpy(d_y_vel_old, h_y_vel, sizeof(double) * PARTICLE_NUM,
    hipMemcpyHostToDevice);
    if (result != hipSuccess) {
        cerr << "Could not copy the y_vel array to the device \n";
        return 0;
    }

    result = hipMemcpy(d_z_vel_old, h_z_vel, sizeof(double) * PARTICLE_NUM,
    hipMemcpyHostToDevice);
    if (result != hipSuccess) {
        cerr << "Could not copy the z_vel array to the device \n";
        return 0;
    }

    for (int iter = 0; iter < DURATION; iter++) {
        // TODO: check time
        // set up the kernel launch parameters
        newState << <(PARTICLE_NUM + THREADS_PER_BLOCK - 1) / THREADS_PER_BLOCK, THREADS_PER_BLOCK >> >(
            PARTICLE_NUM,
            d_masses,
            d_x_pos_old,
            d_y_pos_old,
            d_z_pos_old,
            d_x_vel_old,
            d_y_vel_old,
            d_z_vel_old,
            d_x_pos_new,
            d_y_pos_new,
            d_z_pos_new,
            d_x_vel_new,
            d_y_vel_new,
            d_z_vel_new,
            d_x_acc,
            d_y_acc,
            d_z_acc
        );

        // log results
        result = hipMemcpy(h_x_pos, d_x_pos_new, sizeof(double) * PARTICLE_NUM,
            hipMemcpyDeviceToHost);
        if (result != hipSuccess) {
            cerr << "Could not copy the x_pos array to the host \n";
            return 0;
        }
        result = hipMemcpy(h_y_pos, d_y_pos_new, sizeof(double) * PARTICLE_NUM,
            hipMemcpyDeviceToHost);
        if (result != hipSuccess) {
            cerr << "Could not copy the y_pos array to the host \n";
            return 0;
        }
        result = hipMemcpy(h_z_pos, d_z_pos_new, sizeof(double) * PARTICLE_NUM,
            hipMemcpyDeviceToHost);
        if (result != hipSuccess) {
            cerr << "Could not copy the z_pos array to the host \n";
            return 0;
        }

        for (int particle = 0; particle < PARTICLE_NUM; particle++) {
            cout << h_x_pos[particle] << " "
                << h_y_pos[particle] << " "
                << h_z_pos[particle] << " "
                << endl;
        }
        cout << endl;
        // also velocities?

        // swap the old and new positions and velocities
        swap(d_x_pos_old, d_x_pos_new);
        swap(d_y_pos_old, d_y_pos_new);
        swap(d_z_pos_old, d_z_pos_new);

        swap(d_x_vel_old, d_x_vel_new);
        swap(d_y_vel_old, d_y_vel_new);
        swap(d_z_vel_old, d_z_vel_new);
    }
    
}