#include "hip/hip_runtime.h"
#include "cudaParticleSimulator.cuh"

#if VERSION == 0
    #define EPS 1e-10
    #define G 6.674e-11
    #define STEP_TIME 10.0
#else
    #define EPS 1e-10f
    #define G 6.674e-11f
    #define STEP_TIME 10.0f
#endif

// AoS global FP64
__global__ void newState_0(
    Particle* particles_old,
    Particle* particles_new
) {
    // calculate the index of the current thread
    uint64_t idx = blockIdx.x * blockDim.x + threadIdx.x;

    Particle thread_particle = particles_old[idx];
    double3 acc = make_double3(0.0, 0.0, 0.0);

    // calculate the acceleration of the current particle 
    // cycle through all other particles
    for (uint64_t particle = 0; particle < PARTICLE_NUM; particle++) {
        if (idx != particle) {
            double dx = particles_old[particle].pos.x - thread_particle.pos.x;
            double dy = particles_old[particle].pos.y - thread_particle.pos.y;
            double dz = particles_old[particle].pos.z - thread_particle.pos.z;

            double dist = sqrt(dx * dx + dy * dy + dz * dz + EPS * EPS);
            
            double acc_mod = G * particles_old[particle].mass / (dist * dist);
            acc.x += acc_mod * dx / dist;
            acc.y += acc_mod * dy / dist;
            acc.z += acc_mod * dz / dist;
        }
    }

    // update the velocity and position of the current particle
    particles_new[idx].vel.x = thread_particle.vel.x + acc.x * STEP_TIME;
    particles_new[idx].vel.y = thread_particle.vel.y + acc.y * STEP_TIME;
    particles_new[idx].vel.z = thread_particle.vel.z + acc.z * STEP_TIME;

    particles_new[idx].pos.x = thread_particle.pos.x + thread_particle.vel.x * STEP_TIME + 0.5 * acc.x * STEP_TIME * STEP_TIME;
    particles_new[idx].pos.y = thread_particle.pos.y + thread_particle.vel.y * STEP_TIME + 0.5 * acc.y * STEP_TIME * STEP_TIME;
    particles_new[idx].pos.z = thread_particle.pos.z + thread_particle.vel.z * STEP_TIME + 0.5 * acc.z * STEP_TIME * STEP_TIME;
}


// AoS global FP32 e int
__global__ void newState_1(
    Particle* particles_old,
    Particle* particles_new
) {
    // calculate the index of the current thread
    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    Particle thread_particle = particles_old[idx];
    float3 acc = make_float3(0.0f, 0.0f, 0.0f);

    // calculate the acceleration of the current particle 
    // cycle through all other particles
    for (int particle = 0; particle < PARTICLE_NUM; particle++) {
        if (idx != particle) {
            float dx = particles_old[particle].pos.x - thread_particle.pos.x;
            float dy = particles_old[particle].pos.y - thread_particle.pos.y;
            float dz = particles_old[particle].pos.z - thread_particle.pos.z;

            float dist = sqrtf(dx * dx + dy * dy + dz * dz + EPS * EPS);
            
            float acc_mod = G * particles_old[particle].mass / (dist * dist);
            acc.x += acc_mod * dx / dist;
            acc.y += acc_mod * dy / dist;
            acc.z += acc_mod * dz / dist;
        }
    }

    // update the velocity and position of the current particle
    particles_new[idx].vel.x = thread_particle.vel.x + acc.x * STEP_TIME;
    particles_new[idx].vel.y = thread_particle.vel.y + acc.y * STEP_TIME;
    particles_new[idx].vel.z = thread_particle.vel.z + acc.z * STEP_TIME;

    particles_new[idx].pos.x = thread_particle.pos.x + thread_particle.vel.x * STEP_TIME + 0.5f * acc.x * STEP_TIME * STEP_TIME;
    particles_new[idx].pos.y = thread_particle.pos.y + thread_particle.vel.y * STEP_TIME + 0.5f * acc.y * STEP_TIME * STEP_TIME;
    particles_new[idx].pos.z = thread_particle.pos.z + thread_particle.vel.z * STEP_TIME + 0.5f * acc.z * STEP_TIME * STEP_TIME;
}


// SoA global
__global__ void newState_2(
    const float* masses,
    const float* x_pos_old,
    const float* y_pos_old,
    const float* z_pos_old,
    const float* x_vel_old,
    const float* y_vel_old,
    const float* z_vel_old,
    float* x_pos_new,
    float* y_pos_new,
    float* z_pos_new,
    float* x_vel_new,
    float* y_vel_new,
    float* z_vel_new
) {

    // calculate the index of the current thread
    uint64_t idx = blockIdx.x * blockDim.x + threadIdx.x;

    // set acc to zero
    float x_acc = 0.0f;
    float y_acc = 0.0f;
    float z_acc = 0.0f;

    float thread_x_pos_old = x_pos_old[idx];
    float thread_y_pos_old = y_pos_old[idx];
    float thread_z_pos_old = z_pos_old[idx];

    // calculate the acceleration of the current particle 
    // cycle through all other particles
    for (int particle = 0; particle < PARTICLE_NUM; particle++) {
        if (idx != particle) {
            float dx = x_pos_old[particle] - thread_x_pos_old;
            float dy = y_pos_old[particle] - thread_y_pos_old;
            float dz = z_pos_old[particle] - thread_z_pos_old;

            float dist = sqrtf(dx * dx + dy * dy + dz * dz + EPS * EPS);
            
            float acc_mod = G * masses[particle] / (dist * dist);
            x_acc += acc_mod * dx / dist;
            y_acc += acc_mod * dy / dist;
            z_acc += acc_mod * dz / dist;
        }
    }


    float thread_x_vel_old = x_vel_old[idx];
    float thread_y_vel_old = y_vel_old[idx];
    float thread_z_vel_old = z_vel_old[idx];

    // update the velocity and position of the current particle
    x_vel_new[idx] = thread_x_vel_old + x_acc * STEP_TIME;
    y_vel_new[idx] = thread_y_vel_old + y_acc * STEP_TIME;
    z_vel_new[idx] = thread_z_vel_old + z_acc * STEP_TIME;

    x_pos_new[idx] = thread_x_pos_old + thread_x_vel_old * STEP_TIME + 0.5f * x_acc * STEP_TIME * STEP_TIME;
    y_pos_new[idx] = thread_y_pos_old + thread_y_vel_old * STEP_TIME + 0.5f * y_acc * STEP_TIME * STEP_TIME;
    z_pos_new[idx] = thread_z_pos_old + thread_z_vel_old * STEP_TIME + 0.5f * z_acc * STEP_TIME * STEP_TIME;
}



// each thread loads 1 particle into shared memory
// SoA shared memory
__global__ void newState_3(
    const float* masses,
    const float* x_pos_old,
    const float* y_pos_old,
    const float* z_pos_old,
    const float* x_vel_old,
    const float* y_vel_old,
    const float* z_vel_old,
    float* x_pos_new,
    float* y_pos_new,
    float* z_pos_new,
    float* x_vel_new,
    float* y_vel_new,
    float* z_vel_new
) {

    // calculate the index of the current thread
    uint64_t idx = blockIdx.x * blockDim.x + threadIdx.x;
    uint64_t tidx = threadIdx.x;

    float thread_x_pos_old = x_pos_old[idx];
    float thread_y_pos_old = y_pos_old[idx];
    float thread_z_pos_old = z_pos_old[idx];
    float x_acc = 0.0f;
    float y_acc = 0.0f;
    float z_acc = 0.0f;

    __shared__ float tile_x_pos_old_shared[THREADS_PER_BLOCK];
    __shared__ float tile_y_pos_old_shared[THREADS_PER_BLOCK];
    __shared__ float tile_z_pos_old_shared[THREADS_PER_BLOCK];
    __shared__ float tile_masses_shared[THREADS_PER_BLOCK];

    // iterate over all tiles in PARTICLE_NUM particles
    for (int tile = 0; tile < (PARTICLE_NUM + THREADS_PER_BLOCK - 1) / THREADS_PER_BLOCK; tile++) {
        // load the current tile into shared memory
        // each thread loads the corresponding element
        int shared_idx = tile * THREADS_PER_BLOCK + tidx;
        if (shared_idx < PARTICLE_NUM) {
            tile_x_pos_old_shared[tidx] = x_pos_old[shared_idx];
            tile_y_pos_old_shared[tidx] = y_pos_old[shared_idx];
            tile_z_pos_old_shared[tidx] = z_pos_old[shared_idx];
            tile_masses_shared[tidx] = masses[shared_idx];
        } else {
            // if the shared index is out of bounds, set to zero
            tile_x_pos_old_shared[tidx] = 0.0f;
            tile_y_pos_old_shared[tidx] = 0.0f;
            tile_z_pos_old_shared[tidx] = 0.0f;
            tile_masses_shared[tidx] = 0.0f;
        }

        // synchronize threads to ensure all data is loaded
        __syncthreads();

        // tile all particles in the tile and compute the acceleration
        for (int particle = 0; particle < THREADS_PER_BLOCK; particle++) {
            // check if the particle is within bounds
            if (tile * THREADS_PER_BLOCK + particle < PARTICLE_NUM && idx != tile * THREADS_PER_BLOCK + particle) {
                float dx = tile_x_pos_old_shared[particle] - thread_x_pos_old;
                float dy = tile_y_pos_old_shared[particle] - thread_y_pos_old;
                float dz = tile_z_pos_old_shared[particle] - thread_z_pos_old;

                float dist = sqrtf(dx * dx + dy * dy + dz * dz + EPS * EPS);
                
                float acc_mod = G * tile_masses_shared[particle] / (dist * dist);
                x_acc += acc_mod * dx / dist;
                y_acc += acc_mod * dy / dist;
                z_acc += acc_mod * dz / dist;
            }
        }

        // synchronize threads to ensure all data of the tile has been processed
        __syncthreads();
    }

    float thread_x_vel_old = x_vel_old[idx];
    float thread_y_vel_old = y_vel_old[idx];
    float thread_z_vel_old = z_vel_old[idx];

    // update the velocity and position of the current particle
    x_vel_new[idx] = thread_x_vel_old + x_acc * STEP_TIME;
    y_vel_new[idx] = thread_y_vel_old + y_acc * STEP_TIME;
    z_vel_new[idx] = thread_z_vel_old + z_acc * STEP_TIME;

    x_pos_new[idx] = thread_x_pos_old + thread_x_vel_old * STEP_TIME + 0.5f * x_acc * STEP_TIME * STEP_TIME;
    y_pos_new[idx] = thread_y_pos_old + thread_y_vel_old * STEP_TIME + 0.5f * y_acc * STEP_TIME * STEP_TIME;
    z_pos_new[idx] = thread_z_pos_old + thread_z_vel_old * STEP_TIME + 0.5f * z_acc * STEP_TIME * STEP_TIME;
}