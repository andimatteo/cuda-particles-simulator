#include "hip/hip_runtime.h"
#include "cudaParticleSimulator.cuh"

#ifndef EPS
    #define EPS 1e-10
#endif

#ifndef G
    #define G 6.674e-11
#endif

#ifndef STEP
    #define STEP 0.01
#endif

__global__ void newState(
    const uint64_t particleNum,
    const float64_t* masses,
    const float64_t* x_pos_old,
    const float64_t* y_pos_old,
    const float64_t* z_pos_old,
    const float64_t* x_vel_old,
    const float64_t* y_vel_old,
    const float64_t* z_vel_old,
    float64_t* x_pos_new,
    float64_t* y_pos_new,
    float64_t* z_pos_new,
    float64_t* x_vel_new,
    float64_t* y_vel_new,
    float64_t* z_vel_new,
    float64_t* x_acc,
    float64_t* y_acc,
    float64_t* z_acc
) {

    // calculate the index of the current thread
    uint64_t idx = blockIdx.x * blockDim.x + threadIdx.x;

    // set acc to zero
    x_acc[idx] = 0;
    y_acc[idx] = 0;
    z_acc[idx] = 0;

    // calculate the acceleration of the current particle 
    // cycle through all other particles
    for (uint64_t particle = 0; particle < particleNum; particle++) {
        if (idx != particle) {
            float64_t dx = x_pos_old[particle] - x_pos_old[idx];
            float64_t dy = y_pos_old[particle] - y_pos_old[idx];
            float64_t dz = z_pos_old[particle] - z_pos_old[idx];

            float64_t dist = sqrt(dx * dx + dy * dy + dz * dz + EPS * EPS);
            
            float64_t force = G * masses[particle] / (dist * dist);
            x_acc[idx] += force * dx / dist;
            y_acc[idx] += force * dy / dist;
            z_acc[idx] += force * dz / dist;
        }
    }

    // update the velocity and position of the current particle
    x_vel_new[idx] = x_vel_old[idx] + x_acc[idx] * STEP;
    y_vel_new[idx] = y_vel_old[idx] + y_acc[idx] * STEP;
    z_vel_new[idx] = z_vel_old[idx] + z_acc[idx] * STEP;

    x_pos_new[idx] = x_pos_old[idx] + x_vel_new[idx] * STEP + 0.5 * x_acc[idx] * STEP * STEP;
    y_pos_new[idx] = y_pos_old[idx] + y_vel_new[idx] * STEP + 0.5 * y_acc[idx] * STEP * STEP;
    z_pos_new[idx] = z_pos_old[idx] + z_vel_new[idx] * STEP + 0.5 * z_acc[idx] * STEP * STEP;
}