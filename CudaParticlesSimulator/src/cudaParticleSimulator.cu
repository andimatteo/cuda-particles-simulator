#include "hip/hip_runtime.h"
#include "cudaParticleSimulator.cuh"

#ifndef EPS
    #define EPS 1e-10
#endif

#ifndef G
    #define G 6.674e-11
#endif

#ifndef STEP_TIME
    #define STEP_TIME 0.01
#endif

__global__ void newState(
    const uint64_t particleNum,
    const double* masses,
#if VERSION == 0
    const double* x_pos_old,
    const double* y_pos_old,
    const double* z_pos_old,
    const double* x_vel_old,
    const double* y_vel_old,
    const double* z_vel_old,
    double* x_pos_new,
    double* y_pos_new,
    double* z_pos_new,
    double* x_vel_new,
    double* y_vel_new,
    double* z_vel_new,
    double* x_acc,
    double* y_acc,
    double* z_acc
#else
    const double3* pos_old,
    const double3* vel_old,
    double3* pos_new,
    double3* vel_new,
    double3* acc
#endif
) {

    // calculate the index of the current thread
    uint64_t idx = blockIdx.x * blockDim.x + threadIdx.x;

    // set acc to zero
#if VERSION == 0
    x_acc[idx] = 0;
    y_acc[idx] = 0;
    z_acc[idx] = 0;
#else
    acc[idx] = make_double3(0.0, 0.0, 0.0);
#endif

    // calculate the acceleration of the current particle 
    // cycle through all other particles
    for (uint64_t particle = 0; particle < particleNum; particle++) {
        if (idx != particle) {
#if VERSION == 0
            double dx = x_pos_old[particle] - x_pos_old[idx];
            double dy = y_pos_old[particle] - y_pos_old[idx];
            double dz = z_pos_old[particle] - z_pos_old[idx];
#else
            double dx = pos_old[particle].x - pos_old[idx].x;
            double dy = pos_old[particle].y - pos_old[idx].y;
            double dz = pos_old[particle].z - pos_old[idx].z;
#endif

            double dist = sqrt(dx * dx + dy * dy + dz * dz + EPS * EPS);
            
            double acc_mod = G * masses[particle] / (dist * dist);
#if VERSION == 0
            x_acc[idx] += acc_mod * dx / dist;
            y_acc[idx] += acc_mod * dy / dist;
            z_acc[idx] += acc_mod * dz / dist;
#else
            acc[idx].x += acc_mod * dx / dist;
            acc[idx].y += acc_mod * dy / dist;
            acc[idx].z += acc_mod * dz / dist;
#endif
        }
    }

    // update the velocity and position of the current particle
#if VERSION == 0
    x_vel_new[idx] = x_vel_old[idx] + x_acc[idx] * STEP_TIME;
    y_vel_new[idx] = y_vel_old[idx] + y_acc[idx] * STEP_TIME;
    z_vel_new[idx] = z_vel_old[idx] + z_acc[idx] * STEP_TIME;

    x_pos_new[idx] = x_pos_old[idx] + x_vel_new[idx] * STEP_TIME + 0.5 * x_acc[idx] * STEP_TIME * STEP_TIME;
    y_pos_new[idx] = y_pos_old[idx] + y_vel_new[idx] * STEP_TIME + 0.5 * y_acc[idx] * STEP_TIME * STEP_TIME;
    z_pos_new[idx] = z_pos_old[idx] + z_vel_new[idx] * STEP_TIME + 0.5 * z_acc[idx] * STEP_TIME * STEP_TIME;
#else
    vel_new[idx].x = vel_old[idx].x + acc[idx].x * STEP_TIME;
    vel_new[idx].y = vel_old[idx].y + acc[idx].y * STEP_TIME;
    vel_new[idx].z = vel_old[idx].z + acc[idx].z * STEP_TIME;

    pos_new[idx].x = pos_old[idx].x + vel_new[idx].x * STEP_TIME + 0.5 * acc[idx].x * STEP_TIME * STEP_TIME;
    pos_new[idx].y = pos_old[idx].y + vel_new[idx].y * STEP_TIME + 0.5 * acc[idx].y * STEP_TIME * STEP_TIME;
    pos_new[idx].z = pos_old[idx].z + vel_new[idx].z * STEP_TIME + 0.5 * acc[idx].z * STEP_TIME * STEP_TIME;
#endif
}