#include "hip/hip_runtime.h"
#include "memutils.cuh"

template <typename T>
T* allocateAndCopy(T* array, int length) {
	T* dev_array;
	hipError_t result = hipMalloc((void**)&dev_array, length * sizeof(T));
	if (result != hipSuccess) {
        cerr << "Could not allocate the " << typeid(T).name() << " array \n";
		return 0;
	}
	result = hipMemcpy(dev_array, array, sizeof(T) * length,
		hipMemcpyHostToDevice);
	if (result != hipSuccess) {
		cerr << "Could not copy the " << typeid(T).name() << " array to the device \n";
		return 0;
	}
	return dev_array;
}

template <typename T>
T* allocateAndNull(int length) {
    T* dev_array;
    hipError_t result = hipMalloc((void**)&dev_array, length * sizeof(T));
    if (result != hipSuccess) {
        cerr << "Could not allocate the " << typeid(T).name() << " array \n";
        return 0;
    }
    result = hipMemset(dev_array, 0, sizeof(T) * length);
    if (result != hipSuccess) {
        cerr << "Could not set the " << typeid(T).name() << " array to zero \n";
        return 0;
    }
    return dev_array;
}

template double* allocateAndCopy<double>(double* array, int length);
template double3* allocateAndCopy<double3>(double3* array, int length);
template Particle* allocateAndCopy<Particle>(Particle* array, int length);

template double* allocateAndNull<double>(int length);
template double3* allocateAndNull<double3>(int length);
template Particle* allocateAndNull<Particle>(int length);