#include "memutils.cuh"

double* allocateAndCopy(double* array, int length) {
	double* dev_array;
	hipError_t result = hipMalloc((void**)&dev_array, length * sizeof(double));
	if (result != hipSuccess) {
        cerr << "Could not allocate the double array \n";
		return 0;
	}
	result = hipMemcpy(dev_array, array, sizeof(double) * length,
		hipMemcpyHostToDevice);
	if (result != hipSuccess) {
		cerr << "Could not copy the double array to the device \n";
		return 0;
	}
	return dev_array;
}

double* allocateAndNull(int length) {
    double* dev_array;
    hipError_t result = hipMalloc((void**)&dev_array, length * sizeof(double));
    if (result != hipSuccess) {
        cerr << "Could not allocate the double array \n";
        return 0;
    }
    result = hipMemset(dev_array, 0, sizeof(double) * length);
    if (result != hipSuccess) {
        cerr << "Could not set the double array to zero \n";
        return 0;
    }
    return dev_array;
}